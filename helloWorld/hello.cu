
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {

    printf("Hello world from GPU.\n");
}


int main(void) {

    printf("Hello world from CPU.\n\n");

    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();

    return 0;
}


