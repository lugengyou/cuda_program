
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    int i = threadIdx.x;
    printf("Hello world from GPU by thread %d.\n", i);
}

int main(void) {

    printf("Hello world from CPU.\n\n");

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d: \"%s\"\n", dev, deviceProp.name);

    // 共享内存存储体默认大小
    hipSharedMemConfig pConfig;
    hipDeviceGetSharedMemConfig(&pConfig);
    printf("Shared memory configuration: %d\n", pConfig);

    // helloFromGPU<<<1, 10>>>();
    
    hipDeviceSynchronize();

    hipDeviceReset();

    return 0;
}


